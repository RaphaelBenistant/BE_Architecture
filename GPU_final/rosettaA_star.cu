#include "hip/hip_runtime.h"
#include <list>
#include <algorithm>
#include <iostream>
#include <cstdlib>
#include <chrono>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

using namespace std; 
using namespace std::chrono;

#define SQUARE_SIDE_SIZE 8
#define WALL_PERCENTAGE 0.2         //To avoid no solution, max = 0.4
#define x_start 0                   //min= 0, max = SQUARE_SIDE_SIZE-1
#define y_start 0                   //min= 0, max = SQUARE_SIDE_SIZE-1
#define x_end 3                     //min= 0, max = SQUARE_SIDE_SIZE-1
#define y_end 8                     //min= 0, max = SQUARE_SIDE_SIZE-1

class point {
public:
    point( int a = 0, int b = 0 ) { x = a; y = b; }
    bool operator ==( const point& o ) { return o.x == x && o.y == y; }
    point operator +( const point& o ) { return point( o.x + x, o.y + y ); }
    int x, y;
};
 
class map {
public:
    map() {
        float current_random_value;

        w = h = SQUARE_SIDE_SIZE;
        for( int r = 0; r < h; r++ )
            for( int s = 0; s < w; s++ ){
                if( !( (s ==x_start  && r == y_start) || (s == x_end && r == y_end) )){
                    current_random_value = rand()/(float)RAND_MAX;
                    m[s][r] = current_random_value < WALL_PERCENTAGE ? 1 : 0;
                }
                else m[s][r] = 0;
                
                // cout << "m[" << s << "][" << r <<"] = " << m[s][r] << endl;
            }
            // cout << endl;
    }

    int operator() ( int x, int y ) { return m[x][y]; }
    int m[SQUARE_SIDE_SIZE][SQUARE_SIDE_SIZE];
    int w, h;
};
 
class node {
public:
    bool operator == (const node& o ) { return pos == o.pos; }
    bool operator == (const point& o ) { return pos == o; }
    bool operator < (const node& o ) { return dist + cost < o.dist + o.cost; }
    point pos, parent;
    int dist, cost;
};

//Fonction called from the GPU and executed by the GPU
//--------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------
__device__
bool isValid( point& p ) {
    return ( p.x >-1 && p.y > -1 && p.x < SQUARE_SIDE_SIZE && p.y < SQUARE_SIDE_SIZE );
}

__device__
int dev_calcDist( point& p, point& dev_end){
    // need a better heuristic
    int x = dev_end.x - p.x, y = dev_end.y - p.y;
    return( x * x + y * y );
}

 //If we don't find a node with a cheaper path to the same point then we erase the old one and we return true else we return false and we forget the new path
 __device__
 bool existPoint( point& p, int cost, list<node> dev_closed, list<node> dev_open) {
    list<node>::iterator i;
    i = thrust::find( dev_closed.begin(), dev_closed.end(), p );
    if( i != dev_closed.end() ) {
        if( ( *i ).cost + ( *i ).dist < cost ) return true;
        else { dev_closed.erase( i ); return false; }
    }
    i = thrust::find( dev_open.begin(), dev_open.end(), p );
    if( i != dev_open.end() ) {
        if( ( *i ).cost + ( *i ).dist < cost ) return true;
        else { dev_open.erase( i ); return false; }
    }
    return false;
}
//--------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------

class aStar {
public:
    aStar() {
        neighbours[0] = point( -1, -1 ); neighbours[1] = point(  1, -1 );
        neighbours[2] = point( -1,  1 ); neighbours[3] = point(  1,  1 );
        neighbours[4] = point(  0, -1 ); neighbours[5] = point( -1,  0 );
        neighbours[6] = point(  0,  1 ); neighbours[7] = point(  1,  0 );
    }
 
    int calcDist( point& p ){
        // need a better heuristic
        int x = end.x - p.x, y = end.y - p.y;
        return( x * x + y * y );
    }

    __global__
    void fillOpen(node* dev_n, point* dev_neighbours, int* dev_map, bool* dev_found, list<node> dev_open, list<node> dev_closed) {
        int stepCost, nc, dist;
        point neighbour;

        int i = threadIdx.x + blockIdx.x * blockDim.x;

        //We investigate all neighbours
        // one can make diagonals have different cost
        stepCost = i < 4 ? 1 : 1; //The variable neigbours has the direct neighbours from index 0 to 3 and the diagonal neighbours from index 4 to 7
        neighbour = dev_n->pos + dev_neighbours[i]; //The variable neighbours contains the relative moves from the current position to find the neighbours
        if( neighbour == end ) *dev_found = true;

        if( isValid( neighbour ) && dev_map[neighbour.x, neighbour.y] != 1 ) { //Here we inspect the new position if the position is in the map and the position isn't a wall
            nc = stepCost + dev_n->cost;
            dist = calcDist( neighbour );
            if( !existPoint( neighbour, nc + dist , dev_open, dev_closed) ) { //If we don't have any path to the same point in open or closed where the cost is cheaper, we create a new node in open
                node m;
                m.cost = nc; m.dist = dist;
                m.pos = neighbour; 
                m.parent = dev_n->pos;
                dev_open.push_back( m );
                }
        }
        *dev_found = false;
    }
 
    /*
    You specify a beginning point, an end point, and a map where you want to find the cheapest way.
    It initializes all attributes from the object astar to keep these data in mind.
    We create the first node with parent 0 and current_pos the first position with a cost of zero.
    */
    bool search( point& s, point& e, map& mp ) {

        //Allocate memory in the GPU
        point* dev_neighbours;
        point* dev_end;
        node* dev_n;
        int* dev_map;

        bool* host_found;
        bool* dev_found;

        list<node>* dev_open;
        list<node>* dev_closed;

        hipMalloc( (void**)&dev_neighbours, 8*sizeof(point) ); //Declare the neighbours variable for the GPU
        hipMalloc( (void**)&dev_end, sizeof(point) ); //Declare the end point for the GPU
        hipMalloc( (void**)&dev_map, SQUARE_SIDE_SIZE*SQUARE_SIDE_SIZE*sizeof(point) ); //Declare the end point for the GPU

        //Copy values in the GPU's memory
        hipMemcpy( dev_end, &e, sizeof(point), hipMemcpyHostToDevice );
        hipMemcpy( dev_neighbours, neighbours, 8*sizeof(point), hipMemcpyHostToDevice );
        hipMemcpy( dev_map, mp.m, SQUARE_SIDE_SIZE*SQUARE_SIDE_SIZE*sizeof(int), hipMemcpyHostToDevice );

        node n; end = e; start = s; m = mp;
        n.cost = 0; n.pos = s; n.parent = 0; n.dist = calcDist( s ); 
        open.push_back( n );
        while( !open.empty() ) { //Search stops when all nodes are closed, it means all ways have been inverstigated
            //open.sort();
            node n = open.front(); //FIFO research
            open.pop_front(); //As we investigated the node, we can consider it closed (i.e. investigated)
            closed.push_back( n ); //So we fill the node in closed to keep it in memory
            
            //Declare the current node that will be processed by the GPU
            hipMalloc( (void**)&dev_n, sizeof(node) );
            hipMemcpy( dev_n, &n, sizeof(node), hipMemcpyHostToDevice );

            //Declare the bool result in the GPU that is needed for our stop condition
            hipMalloc( (void**)&dev_found, sizeof(bool) );

            //Create device open and close list
            hipMalloc( (void**)&dev_open, open.size()*sizeof(node) );
            hipMalloc( (void**)&dev_closed, closed.size()*sizeof(node) );
            hipMemcpy( dev_open, open, open.size()*sizeof(node), hipMemcpyHostToDevice );
            hipMemcpy( dev_closed, closed, closed.size()*sizeof(node), hipMemcpyHostToDevice );

            fillOpen<<<1,8>>>( dev_n, dev_neighbours, dev_map, dev_found, dev_open, dev_closed);
            
            //We update CPU's open and closed lists using the one that were modified by the kernel
            hipMemcpy( open, dev_open, dev_open.size()*sizeof(node), hipMemcpyDeviceToHost );
            hipMemcpy( closed, dev_closed, dev_closed.size()*sizeof(node), hipMemcpyDeviceToHost );
            
            //We free GPU's open and closed lists
            hipFree(dev_open);
            hipFree(dev_closed);

            hipMemcpy( host_found, dev_found, sizeof(bool), hipMemcpyDeviceToHost );
            if( *host_found ){
                //Free GPU's memory
                hipFree(dev_found);
                hipFree(dev_n);
                hipFree(dev_end);
                hipFree(dev_neighbours);
                hipFree(dev_map);
                return true;
             }
             hipFree(dev_n);
             hipFree(dev_found);
        }
        //Free GPU's memory
        hipFree(dev_end);
        hipFree(dev_neighbours);
        hipFree(dev_map);
        return false;
    }
 
    /*
    Recreate the path from the closed list containing all the nodes that leads to the solution
    */
    int path( list<point>& path ) {
        path.push_front( end ); //We last nodes first so at the end, the path list will be in the right order
        int cost = 1 + closed.back().cost; //We consider the last move to the end to cost 1 ????
        path.push_front( closed.back().pos );
        point parent = closed.back().parent;
 
        for( list<node>::reverse_iterator i = closed.rbegin(); i != closed.rend(); i++ ) { //We go through the entire close node list till we reach the start point
            if( ( *i ).pos == parent && !( ( *i ).pos == start ) ) {
                path.push_front( ( *i ).pos );
                parent = ( *i ).parent;
            }
        }
        path.push_front( start );
        return cost;
    }
 
    map m; point end, start;
    point neighbours[8];
    list<node> open;
    list<node> closed;

};
 

int main( int argc, char* argv[] ) {
    map m;
    point s(x_start,y_start), e(x_end,y_end); //s is the start e is the end
    aStar as;

    //Start point to measure executions time
    auto start = high_resolution_clock::now();

    if( as.search( s, e, m ) ) {
        list<point> path;
        int c = as.path( path );
        for( int y = -1; y < SQUARE_SIDE_SIZE+1; y++ ) {
            for( int x = -1; x < SQUARE_SIDE_SIZE+1; x++ ) {
                if( x < 0 || y < 0 || x > SQUARE_SIDE_SIZE-1 || y > SQUARE_SIDE_SIZE-1 || m( x, y ) == 1 )
                    cout << "w";
                else {
                    if( find( path.begin(), path.end(), point( x, y ) )!= path.end() )
                        cout << "x";
                    else cout << ".";
                }
            }
            cout << "\n";
        }
 
        cout << "\nPath cost " << c << ": ";
        for( list<point>::iterator i = path.begin(); i != path.end(); i++ ) {
            cout<< "(" << ( *i ).x << ", " << ( *i ).y << ") ";
        }
    }
    cout << "\n\n";

    // Stop point to measure executions time
    auto stop = high_resolution_clock::now();

    // Display execution time
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "CPU execution time = " << duration.count() << " microseconds" <<endl;

    return 0;
}
